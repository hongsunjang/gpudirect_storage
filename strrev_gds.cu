#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <chrono>
#include <iomanip>
#include <vector>
#include "cufile.h"
#include <cstring>
#include <cerrno>
//#include "cufile_sample_utils.h"

#define GB(x) ((x)*1024L*1024L*1024L)
#define MB(x) ((x)*1024L*1024L)
#define KB(x) ((x)*1024L)

// POSIX
template<class T,
	typename std::enable_if<std::is_integral<T>::value, std::nullptr_t>::type = nullptr>
std::string cuFileGetErrorString(T status) {
	status = std::abs(status);
	return IS_CUFILE_ERR(status) ?
		std::string(CUFILE_ERRSTR(status)) : std::string(std::strerror(status));
}

// CUfileError_t
template<class T,
	typename std::enable_if<!std::is_integral<T>::value, std::nullptr_t>::type = nullptr>
std::string cuFileGetErrorString(T status) {
	std::string errStr = cuFileGetErrorString(static_cast<int>(status.err));
	if (IS_CUDA_ERR(status))
		errStr.append(".").append(GetCuErrorString(status.cu_err));
	return errStr;
}

template <typename T>
struct aligned_allocator
{
  using value_type = T;
  T* allocate(std::size_t num)
  {
    void* ptr = nullptr;
    if (posix_memalign(&ptr,4096,num*sizeof(T)))
      throw std::bad_alloc();
    return reinterpret_cast<T*>(ptr);
  }
  void deallocate(T* p, std::size_t num)
  {
    free(p);
  }
};

__global__ void hello(char *str) {
	printf("Hello World!\n");
	printf("buf: %s\n", str);
}

int main(int argc, char *argv[])
{
	int fd;
	int ret;

	char *gpumem_buf;

	CUfileDescr_t cf_desc; 
	CUfileHandle_t cf_handle;
	CUfileError_t status;
	status = cuFileDriverOpen();
	if (status.err != CU_FILE_SUCCESS) {
			std::cerr << "cufile driver open error: "<<std::endl;
			return -1;
	}

	fd = open(argv[1], O_RDWR | O_DIRECT);
	if (fd == -1) {
        perror("open");
        return 1;
    }
	struct stat st;
    if (fstat(fd, &st) == -1) {
        perror("fstat");
        close(fd);
        return 1;
    }	
	

	int blksize = 512;
	//uint file_size_in_bytes = ((st.st_size -1)/blksize + 1) *blksize ;
	uint file_size_in_bytes = GB(1) ;
	
	// Print the file size in bytes
    std::cout << "File size: " << (double)(file_size_in_bytes) / 1024 / 1024 << " MB" << std::endl;
	
	std::cout << "Done\n";
	hipMalloc(&gpumem_buf, file_size_in_bytes);
	
	off_t file_offset = 0;
	off_t mem_offset = 0;
	
	memset((void*)&cf_desc, 0, sizeof(CUfileDescr_t));
	cf_desc.handle.fd = fd;
	cf_desc.type = CU_FILE_HANDLE_TYPE_OPAQUE_FD;

	status = cuFileHandleRegister(&cf_handle, &cf_desc);
	cuFileBufRegister((char*)gpumem_buf, file_size_in_bytes, 0);

	std::cout << "Read starts..." << std::endl;	
	std::chrono::high_resolution_clock::time_point read_start = std::chrono::high_resolution_clock::now();

	ret = cuFileRead(cf_handle, (char*)gpumem_buf, file_size_in_bytes, file_offset, mem_offset);
	if (ret < 0) {
		printf("cuFileRead failed : %d\n", ret); 
        close(fd);
        return 1;
	}
	close(fd);
	
	std::chrono::high_resolution_clock::time_point read_end = std::chrono::high_resolution_clock::now();
	ulong read_time = std::chrono::duration_cast<std::chrono::microseconds>(read_end - read_start).count();
	
	std::cout << "Read ends...\n" << std::endl;	

    double read_microsec_duration = (double) read_time;
	double read_millisec_duration = read_microsec_duration / 1e3;
	
	std::cout << "CPU: " << read_millisec_duration << " ms"
		<< std::setprecision(6) << std::fixed << "\n";
		
	std::cout << "Throughput: " << (double)(file_size_in_bytes) / GB(1)/ (read_millisec_duration*1e3) << " GB/s"
		<< std::setprecision(6) << std::fixed << "\n";


	//strrev<<<1,1>>>(gpumem_buf, gpu_len);
	std::vector<char, aligned_allocator<char>> system_buf(file_size_in_bytes);
	hipMemcpy(&system_buf[0], gpumem_buf, file_size_in_bytes, hipMemcpyDeviceToHost);
	std::cout << "Done\n";
	fd = open(argv[2], O_RDWR | O_DIRECT| O_CREAT, 0644);
	if (fd == -1) {
        perror("open");
        return 1;
    }

	

	std::cout << "Write starts..." << std::endl;	
	std::chrono::high_resolution_clock::time_point write_start = std::chrono::high_resolution_clock::now();

	ret = pwrite(fd, (void*)&system_buf[0], file_size_in_bytes, 0);
	if (ret == -1) {
		std::cout << "P2P: write() failed, err: " << ret << ", "<< strerror(errno) << ", line: " << __LINE__ << std::endl;
		return EXIT_FAILURE;
	}	

	/*
	cf_desc.handle.fd = fd;
	cuFileHandleRegister(&cf_handle, &cf_desc);

	ret = cuFileWrite(cf_handle, (char*)gpumem_buf, file_size_in_bytes, file_offset, mem_offset);
	if (ret < 0) {
		printf("cuFileWrite failed : %d\n", ret); 
		close(fd);
        return 1;
	}
	*/	
	std::chrono::high_resolution_clock::time_point write_end = std::chrono::high_resolution_clock::now();
	ulong write_time = std::chrono::duration_cast<std::chrono::microseconds>(write_end - write_start).count();
	
	std::cout << "Write ends...\n" << std::endl;	

    double write_microsec_duration = (double) write_time;
	double write_millisec_duration = write_microsec_duration / 1e3;
	
	std::cout << "CPU: " << write_millisec_duration << " ms"
		<< std::setprecision(6) << std::fixed << "\n";
		
	std::cout << "Throughput: " << (double)(file_size_in_bytes) / GB(1) / (write_millisec_duration*1000) << " GB/s"
		<< std::setprecision(6) << std::fixed << "\n";

	//printf("%s\n", system_buf);
	printf("See also %s\n", argv[2]);

	cuFileBufDeregister((char*)gpumem_buf);

	hipFree(gpumem_buf);

	cuFileDriverClose();
}
